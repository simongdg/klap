#include "hip/hip_runtime.h"
#include "common.h"

__inline__ __device__ void
SSSP_54(unsigned int *outgoing_55, int SourceSize_73, int OffsetEnd_74,
        int ObjectSize_75, int Stride_76, unsigned int *dist_p_56,
        unsigned int *dist_57, unsigned int *srcsrc_58, int SourceSize_77,
        int OffsetEnd_78, unsigned int *psrc_59, unsigned int *edgessrcdst_60,
        unsigned int *edgessrcwt_61, bool *changed_62, unsigned int numNodes_63,
        unsigned int numEdges_64) {

  unsigned int tid_79 = threadIdx.x;
  for (int nn_65 = 0; (nn_65 < ObjectSize_75); nn_65 += Stride_76) {
    if ((nn_65 + threadIdx.x < SourceSize_77) &&
        (nn_65 + (blockIdx.x * SourceSize_77 + threadIdx.x) < OffsetEnd_78)) {
      unsigned int neighborsize_66 = outgoing_55[nn_65];
      bool local_change = false;
      for (unsigned int ii_67 = 0; (ii_67 < neighborsize_66); ++ii_67) {
        bool ll_change = false;
        if ((srcsrc_58[nn_65] < numNodes_63)) {
          unsigned int edge_68 = (psrc_59[srcsrc_58[nn_65]] + ii_67);
          if ((edge_68 && (edge_68 < (numEdges_64 + 1)))) {
            unsigned int dst_69 = edgessrcdst_60[edge_68];
            unsigned int wt_70 = edgessrcwt_61[edge_68];
            if (((dst_69 >= numNodes_63) || (wt_70 >= 1000000000))) {
              ll_change = false;
            } else {
              unsigned int altdist_71 = (dist_p_56[nn_65] + wt_70);
              if ((altdist_71 < dist_57[dst_69])) {
                unsigned int olddist_72 =
                    atomicMin(&dist_57[dst_69], altdist_71);
                if ((altdist_71 < olddist_72)) {
                  ll_change = true;
                }
              } else {
                ll_change = false;
              }
            }
          }
        }
        if (ll_change) {
          local_change = true;
        }
      }
      if (local_change) {
        *changed_62 = true;
      }
    }
  }
}

__global__ void /*SSSP_36*/ drelax(
    unsigned int *outgoing_37, int ObjectSize_80, int ObjectSize_81,
    int SourceSize_82, unsigned int *dist_p_38, int ObjectSize_83,
    unsigned int *dist_39, unsigned int *srcsrc_40, int ObjectSize_84,
    int SourceSize_85, unsigned int *psrc_41, unsigned int *edgessrcdst_42,
    unsigned int *edgessrcwt_43, bool *changed_44, unsigned int numNodes_45,
    unsigned int numEdges_46) {

  unsigned int blockID_86 = blockIdx.x;
  int p_47 = blockDim.x;
  int x_size_48 = ObjectSize_81;
  int tile_49 = ((((x_size_48 + p_47) - 1)) / p_47);

  unsigned int *part_outgoing_51 = outgoing_37 + (blockIdx.x * ObjectSize_80);
  unsigned int *part_dist_52 = dist_p_38 + (blockIdx.x * ObjectSize_83);
  unsigned int *part_srcsrc_53 = srcsrc_40 + (blockIdx.x * ObjectSize_84);

  SSSP_54(part_outgoing_51 + (0 + (threadIdx.x * 1)), x_size_48, SourceSize_82,
          (p_47 * tile_49), p_47, part_dist_52 + (0 + (threadIdx.x * 1)),
          dist_39, part_srcsrc_53 + (0 + (threadIdx.x * 1)), x_size_48,
          SourceSize_85, psrc_41, edgessrcdst_42, edgessrcwt_43, changed_44,
          numNodes_45, numEdges_46);

  __syncthreads();
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void SSSP_18(unsigned int *outgoing_19, int ObjectSize_87,
             unsigned int *dist_p_20, unsigned int *dist_21,
             unsigned int *srcsrc_22, unsigned int *psrc_23,
             unsigned int *edgessrcdst_24, unsigned int *edgessrcwt_25,
             bool *changed_26, unsigned int numNodes_27,
             unsigned int numEdges_28) {

  int p_29 = TGM_TEMPLATE_0;
  int x_size_30 = ObjectSize_87;
  int tile_31 = ((((x_size_30 + p_29) - 1)) / p_29);

  unsigned int *part_outgoing_33 = outgoing_19;
  unsigned int *part_dist_34 = dist_p_20;
  unsigned int *part_srcsrc_35 = srcsrc_22;

  dim3 dimBlock(TGM_TEMPLATE_1);
  dim3 dimGrid(p_29);

  /*SSSP_36*/ drelax << <dimGrid, dimBlock>>>
      (part_outgoing_33, tile_31, tile_31, x_size_30, part_dist_34, tile_31,
       dist_21, part_srcsrc_35, tile_31, x_size_30, psrc_23, edgessrcdst_24,
       edgessrcwt_25, changed_26, numNodes_27, numEdges_28);
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void SSSP_tangram(unsigned int *dist_2, unsigned int *outgoing_3,
                  unsigned int *srcsrc_4, unsigned int *psrc_5,
                  unsigned int *edgessrcdst_6, unsigned int *edgessrcwt_7,
                  bool *changed_8, unsigned int numNodes_9,
                  unsigned int numEdges_10) {

  unsigned int *T_dist_11 = dist_2;
  unsigned int *T_dist_p_12 = dist_2;
  unsigned int *T_outgoing_13 = outgoing_3;
  unsigned int *T_srcsrc_14 = srcsrc_4;
  unsigned int *T_psrc_15 = psrc_5;
  unsigned int *T_edgessrcdst_16 = edgessrcdst_6;
  unsigned int *T_edgessrcwt_17 = edgessrcwt_7;

  SSSP_18<TGM_TEMPLATE_0, TGM_TEMPLATE_1>(
      T_outgoing_13, numNodes_9, T_dist_p_12, T_dist_11, T_srcsrc_14, T_psrc_15,
      T_edgessrcdst_16, T_edgessrcwt_17, changed_8, numNodes_9, numEdges_10);
}

void launch_kernel(unsigned int nb, unsigned int nt, foru *dist, Graph graph,
                   bool *changed) {
  SSSP_tangram<32, 128>((unsigned int *)dist, graph.getNoutGoing(),
                        graph.getSrcsrc(), graph.getPsrc(),
                        graph.getEdgessrcdst(), graph.getEdgessrcwt(), changed,
                        graph.getNnodes(), graph.getNedges());
}
