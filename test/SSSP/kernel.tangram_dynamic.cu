#include "hip/hip_runtime.h"
#include "common.h"

__inline__ __device__ void
SSSP_93(unsigned int *outgoing_94, int SourceSize_114, int OffsetEnd_115,
        int ObjectSize_116, int Stride_117, unsigned int *dist_p_95,
        unsigned int *dist_96, unsigned int *srcsrc_97, int SourceSize_118,
        int OffsetEnd_119, unsigned int *psrc_98, unsigned int *edgessrcdst_99,
        unsigned int *edgessrcwt_100, bool *changed_101,
        unsigned int numNodes_102, unsigned int numEdges_103) {

  unsigned int tid_120 = threadIdx.x;
  for (int nn_104 = 0; (nn_104 < ObjectSize_116); nn_104 += Stride_117) {
    if ((nn_104 + threadIdx.x < SourceSize_118) &&
        (nn_104 + (blockIdx.x * SourceSize_118 + threadIdx.x) <
         OffsetEnd_119)) {
      unsigned int neighborsize_105 = outgoing_94[nn_104];
      bool local_change_106 = false;
      for (unsigned int ii_107 = 0; (ii_107 < neighborsize_105); ++ii_107) {
        bool ll_change_108 = false;
        if ((srcsrc_97[nn_104] < numNodes_102)) {
          unsigned int edge_109 = (psrc_98[srcsrc_97[nn_104]] + ii_107);
          if ((edge_109 && (edge_109 < (numEdges_103 + 1)))) {
            unsigned int dst_110 = edgessrcdst_99[edge_109];
            unsigned int wt_111 = edgessrcwt_100[edge_109];
            if (((dst_110 >= numNodes_102) || (wt_111 >= 1000000000))) {
              ll_change_108 = false;
            } else {
              unsigned int altdist_112 = (dist_p_95[nn_104] + wt_111);
              if ((altdist_112 < dist_96[dst_110])) {
                unsigned int olddist_113 =
                    atomicMin(&dist_96[dst_110], altdist_112);
                if ((altdist_112 < olddist_113)) {
                  ll_change_108 = true;
                }
              } else {
                ll_change_108 = false;
              }
            }
          }
        }
        if ((ll_change_108)) {
          local_change_106 = true;
        }
      }
      if ((local_change_106)) {
        *changed_101 = true;
      }
    }
  }
}

__global__ void SSSP_75(unsigned int *outgoing_76, int ObjectSize_121,
                        int ObjectSize_122, int SourceSize_123,
                        unsigned int *dist_p_77, int ObjectSize_124,
                        unsigned int *dist_78, unsigned int *srcsrc_79,
                        int ObjectSize_125, int SourceSize_126,
                        unsigned int *psrc_80, unsigned int *edgessrcdst_81,
                        unsigned int *edgessrcwt_82, bool *changed_83,
                        unsigned int numNodes_84, unsigned int numEdges_85) {

  unsigned int tid_127 = blockIdx.x;
  int p_86 = blockDim.x;
  int x_size_87 = ObjectSize_122;
  int tile_88 = ((((x_size_87 + p_86) - 1)) / p_86);

  unsigned int *part_outgoing_90 = outgoing_76 + (blockIdx.x * ObjectSize_121);
  unsigned int *part_dist_91 = dist_p_77 + (blockIdx.x * ObjectSize_124);
  unsigned int *part_srcsrc_92 = srcsrc_79 + (blockIdx.x * ObjectSize_125);

  SSSP_93(part_outgoing_90 + (0 + (threadIdx.x * 1)), x_size_87, SourceSize_123,
          (p_86 * tile_88), p_86, part_dist_91 + (0 + (threadIdx.x * 1)),
          dist_78, part_srcsrc_92 + (0 + (threadIdx.x * 1)), x_size_87,
          SourceSize_126, psrc_80, edgessrcdst_81, edgessrcwt_82, changed_83,
          numNodes_84, numEdges_85);

  __syncthreads();
}

__inline__ __device__ void
SSSP_54(unsigned int *outgoing_55, int SourceSize_128, int OffsetEnd_129,
        int ObjectSize_130, int Stride_131, unsigned int *dist_p_56,
        unsigned int *dist_57, unsigned int *srcsrc_58, int SourceSize_132,
        int OffsetEnd_133, unsigned int *psrc_59, unsigned int *edgessrcdst_60,
        unsigned int *edgessrcwt_61, bool *changed_62, unsigned int numNodes_63,
        unsigned int numEdges_64) {

  unsigned int tid_134 = threadIdx.x;
  for (int nn_65 = 0; (nn_65 < ObjectSize_130); nn_65 += Stride_131) {
    if ((nn_65 + threadIdx.x < SourceSize_132) &&
        (nn_65 + (blockIdx.x * SourceSize_132 + threadIdx.x) < OffsetEnd_133)) {
      unsigned int neighborsize_66 = outgoing_55[nn_65];
      bool local_change_67 = false;
      for (unsigned int ii_68 = 0; (ii_68 < neighborsize_66); ++ii_68) {
        bool ll_change_69 = false;
        if ((srcsrc_58[nn_65] < numNodes_63)) {
          unsigned int edge_70 = (psrc_59[srcsrc_58[nn_65]] + ii_68);
          if ((edge_70 && (edge_70 < (numEdges_64 + 1)))) {
            unsigned int dst_71 = edgessrcdst_60[edge_70];
            unsigned int wt_72 = edgessrcwt_61[edge_70];
            if (((dst_71 >= numNodes_63) || (wt_72 >= 1000000000))) {
              ll_change_69 = false;
            } else {
              unsigned int altdist_73 = (dist_p_56[nn_65] + wt_72);
              if ((altdist_73 < dist_57[dst_71])) {
                unsigned int olddist_74 =
                    atomicMin(&dist_57[dst_71], altdist_73);
                if ((altdist_73 < olddist_74)) {
                  ll_change_69 = true;
                }
              } else {
                ll_change_69 = false;
              }
            }
          }
        }
        if ((ll_change_69)) {
          local_change_67 = true;
        }
      }
      if ((local_change_67)) {
        *changed_62 = true;
      }
    }
  }
}

__global__ void /*SSSP_36*/ drelax(
    unsigned int *outgoing_37, int ObjectSize_135, int ObjectSize_136,
    int SourceSize_137, unsigned int *dist_p_38, int ObjectSize_138,
    unsigned int *dist_39, unsigned int *srcsrc_40, int ObjectSize_139,
    int SourceSize_140, unsigned int *psrc_41, unsigned int *edgessrcdst_42,
    unsigned int *edgessrcwt_43, bool *changed_44, unsigned int numNodes_45,
    unsigned int numEdges_46) {

  unsigned int blockID_141 = blockIdx.x;
  int p_47 = blockDim.x;
  int x_size_48 = ObjectSize_136;
  int tile_49 = ((((x_size_48 + p_47) - 1)) / p_47);

  unsigned int *part_outgoing_51 = outgoing_37 + (blockIdx.x * ObjectSize_135);
  unsigned int *part_dist_52 = dist_p_38 + (blockIdx.x * ObjectSize_138);
  unsigned int *part_srcsrc_53 = srcsrc_40 + (blockIdx.x * ObjectSize_139);
  /*DYNAMIC*/
  //if(threadIdx.x == 0 && blockIdx.x == 0)
  //  printf("tile = %d \n", tile_49);
  if ((tile_49 > 1023)) {
    //printf("Dynamic\n");
    int tile_dyn = ((((tile_49 + gridDim.x) - 1)) / gridDim.x);

    int blockBound = (blockIdx.x < (gridDim.x - 1))
                         ? x_size_48
                         : (SourceSize_137 - (blockIdx.x * x_size_48));

    int OffsetEnd = (threadIdx.x < (blockDim.x - 1))
                        ? tile_49
                        : (blockBound - (threadIdx.x * tile_49));

    /*
       SSSP_75(unsigned int *outgoing_76, int ObjectSize_121,
               int ObjectSize_122, int SourceSize_123,
               unsigned int *dist_p_77, int ObjectSize_124,
               unsigned int *dist_78, unsigned int *srcsrc_79,
               int ObjectSize_125, int SourceSize_126,
               unsigned int *psrc_80, unsigned int *edgessrcdst_81,
               unsigned int *edgessrcwt_82, bool *changed_83,
               unsigned int numNodes_84, unsigned int numEdges_85) {
    */

    SSSP_75 << <gridDim.x, blockDim.x>>>
        (part_outgoing_51 + (0 + (threadIdx.x * tile_49)),
         tile_dyn /*x_size_48*/, tile_dyn,
         OffsetEnd /*SourceSize_137, (p_47 * tile_49)*/,
         part_dist_52 + (0 + (threadIdx.x * tile_49)), tile_dyn, dist_39,
         part_srcsrc_53 + (0 + (threadIdx.x * tile_49)), tile_dyn /*x_size_48*/,
         OffsetEnd, psrc_41, edgessrcdst_42, edgessrcwt_43, changed_44,
         numNodes_45, numEdges_46);

    hipDeviceSynchronize();

  } else {

    SSSP_54(part_outgoing_51 + (0 + (threadIdx.x * 1)), x_size_48,
            SourceSize_137, (p_47 * tile_49), p_47,
            part_dist_52 + (0 + (threadIdx.x * 1)), dist_39,
            part_srcsrc_53 + (0 + (threadIdx.x * 1)), x_size_48, SourceSize_140,
            psrc_41, edgessrcdst_42, edgessrcwt_43, changed_44, numNodes_45,
            numEdges_46);

    __syncthreads();
  }
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void SSSP_18(unsigned int *outgoing_19, int ObjectSize_142,
             unsigned int *dist_p_20, unsigned int *dist_21,
             unsigned int *srcsrc_22, unsigned int *psrc_23,
             unsigned int *edgessrcdst_24, unsigned int *edgessrcwt_25,
             bool *changed_26, unsigned int numNodes_27,
             unsigned int numEdges_28) {

  int p_29 = TGM_TEMPLATE_0;
  int x_size_30 = ObjectSize_142;
  int tile_31 = ((((x_size_30 + p_29) - 1)) / p_29);

  unsigned int *part_outgoing_33 = outgoing_19;
  unsigned int *part_dist_34 = dist_p_20;
  unsigned int *part_srcsrc_35 = srcsrc_22;

  dim3 dimBlock(TGM_TEMPLATE_1);
  dim3 dimGrid(p_29);
  /*SSSP_36*/ drelax << <dimGrid, dimBlock>>>
      (part_outgoing_33, tile_31, tile_31, x_size_30, part_dist_34, tile_31,
       dist_21, part_srcsrc_35, tile_31, x_size_30, psrc_23, edgessrcdst_24,
       edgessrcwt_25, changed_26, numNodes_27, numEdges_28);
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void SSSP_tangram(unsigned int *dist_2, unsigned int *outgoing_3,
                  unsigned int *srcsrc_4, unsigned int *psrc_5,
                  unsigned int *edgessrcdst_6, unsigned int *edgessrcwt_7,
                  bool *changed_8, unsigned int numNodes_9,
                  unsigned int numEdges_10) {

  unsigned int *T_dist_11 = dist_2;
  unsigned int *T_dist_p_12 = dist_2;
  unsigned int *T_outgoing_13 = outgoing_3;
  unsigned int *T_srcsrc_14 = srcsrc_4;
  unsigned int *T_psrc_15 = psrc_5;
  unsigned int *T_edgessrcdst_16 = edgessrcdst_6;
  unsigned int *T_edgessrcwt_17 = edgessrcwt_7;

  SSSP_18<TGM_TEMPLATE_0, TGM_TEMPLATE_1>(
      T_outgoing_13, numNodes_9, T_dist_p_12, T_dist_11, T_srcsrc_14, T_psrc_15,
      T_edgessrcdst_16, T_edgessrcwt_17, changed_8, numNodes_9, numEdges_10);
}

void launch_kernel(unsigned int nb, unsigned int nt, foru *dist, Graph graph,
                   bool *changed) {
  SSSP_tangram<2, 2>((unsigned int *)dist, graph.getNoutGoing(),
                       graph.getSrcsrc(), graph.getPsrc(),
                       graph.getEdgessrcdst(), graph.getEdgessrcwt(), changed,
                       graph.getNnodes(), graph.getNedges());
}
