#include "hip/hip_runtime.h"
#include "common.h"

__inline__ __device__ void GC_61(int *boundaryListD_62, int SourceSize_71,
                                 int OffsetEnd_72, int ObjectSize_73,
                                 int Stride_74, int *conflictD_63,
                                 int SourceSize_75, int SourceSize_76,
                                 int OffsetEnd_77, int *adjacentListD_64,
                                 int SourceSize_78, int OffsetEnd_79,
                                 int *colors_65, const int maxDegree_66) {

  unsigned int tid_80 = threadIdx.x;
  int i_67;
  for (int idx_68 = 0; (idx_68 < ObjectSize_73); idx_68 += Stride_74) {
    if ((idx_68 + threadIdx.x < SourceSize_71) &&
        (idx_68 + (blockIdx.x * SourceSize_71 + threadIdx.x) < OffsetEnd_72)) {
      //if ((idx_68 + threadIdx.x < SourceSize_76) &&
      //    (idx_68 + (blockIdx.x * SourceSize_76 + threadIdx.x) <
      //     OffsetEnd_77)) {
        i_67 = boundaryListD_62[idx_68];
        conflictD_63[idx_68] = 0;
        for (int k_69 = 0; (k_69 < maxDegree_66); ++k_69) {
          int j_70 = adjacentListD_64[((i_67 * maxDegree_66) + k_69)];
          if (((j_70 < i_67) && ((colors_65[i_67] == colors_65[j_70])))) {
            if (blockIdx.x * blockDim.x + threadIdx.x < SourceSize_75) {
              conflictD_63[idx_68] = (i_67 + 1);
              colors_65[i_67] = 0;
            };
          }
        }
      //}
    }
  }
}

__global__ void GC_49(int *boundaryListD_50, int ObjectSize_81,
                                 int ObjectSize_82, int SourceSize_83,
                                 int *conflictD_51, int ObjectSize_84,
                                 int SourceSize_85, int *adjacentListD_52,
                                 int SourceSize_86, int OffsetEnd_87,
                                 int *colors_53, const int maxDegree_54) {

  unsigned int tid_88 = blockIdx.x;
  int p_55 = blockDim.x;
  int x_size_56 = ObjectSize_82;
  int tile_57 = ((((x_size_56 + p_55) - 1)) / p_55);

  int *part_boundaryList_59 = boundaryListD_50 + (blockIdx.x * ObjectSize_81);
  int *part_conflictD_60 = conflictD_51 + (blockIdx.x * ObjectSize_84);


  GC_61(part_boundaryList_59 + (0 + (threadIdx.x * 1)), x_size_56,
        SourceSize_83, (p_55 * tile_57), p_55,
        part_conflictD_60 + (0 + (threadIdx.x * 1)), x_size_56, x_size_56,
        SourceSize_85, adjacentListD_52, SourceSize_86, OffsetEnd_87, colors_53,
        maxDegree_54);

  __syncthreads();
}


__inline__ __device__ void GC_39(int *boundaryListD_40, int SourceSize_89,
                                 int OffsetEnd_90, int ObjectSize_91,
                                 int Stride_92, int *conflictD_41,
                                 int SourceSize_93, int SourceSize_94,
                                 int OffsetEnd_95, int *adjacentListD_42,
                                 int SourceSize_96, int OffsetEnd_97,
                                 int *colors_43, const int maxDegree_44) {

  unsigned int tid_98 = threadIdx.x;
  int i_45;
  for (int idx_46 = 0; (idx_46 < ObjectSize_91); idx_46 += Stride_92) {
    if ((idx_46 + threadIdx.x < SourceSize_89) &&
        (idx_46 + (blockIdx.x * SourceSize_89 + threadIdx.x) < OffsetEnd_90)) {
      //if ((idx_46 + threadIdx.x < SourceSize_94) &&
      //    (idx_46 + (blockIdx.x * SourceSize_94 + threadIdx.x) <
      //     OffsetEnd_95)) {
        i_45 = boundaryListD_40[idx_46];
        conflictD_41[idx_46] = 0;
        for (int k_47 = 0; (k_47 < maxDegree_44); ++k_47) {
          int j_48 = adjacentListD_42[((i_45 * maxDegree_44) + k_47)];
          if (((j_48 < i_45) && ((colors_43[i_45] == colors_43[j_48])))) {
            if (blockIdx.x * blockDim.x + threadIdx.x < SourceSize_93) {
              conflictD_41[idx_46] = (i_45 + 1);
              colors_43[i_45] = 0;
            }
          }
        }
      //}
    }
  }
}

__global__ void GC_27(int *boundaryListD_28, int ObjectSize_99,
                      int ObjectSize_100, int SourceSize_101, int *conflictD_29,
                      int ObjectSize_102, int SourceSize_103,
                      int *adjacentListD_30, int SourceSize_104,
                      int OffsetEnd_105, int SourceSize_106, int OffsetEnd_107,
                      int *colors_31, const int maxDegree_32) {

  unsigned int blockID_108 = blockIdx.x;
  int p_33 = blockDim.x;
  int x_size_34 = ObjectSize_100;
  int tile_35 = ((((x_size_34 + p_33) - 1)) / p_33);

  int *part_boundaryList_37 = boundaryListD_28 + (blockIdx.x * ObjectSize_99);
  int *part_conflictD_38 = conflictD_29 + (blockIdx.x * ObjectSize_102);
  /*DYNAMIC*/
  //if(threadIdx.x == 0 && blockIdx.x == 0)
  //  printf("tile = %d \n", tile_35);
  if ((tile_35 > 31)) {

   int tile_dyn = ((((tile_35 + gridDim.x) - 1)) / gridDim.x);

    int blockBound = (blockIdx.x < (gridDim.x - 1))
                         ? x_size_34
                         : (SourceSize_101 - (blockIdx.x * x_size_34));

    int OffsetEnd = (threadIdx.x < (blockDim.x - 1))
                        ? tile_35
                        : (blockBound - (threadIdx.x * tile_35));

/*   GC_49(int *boundaryListD_50, int ObjectSize_81,
         int ObjectSize_82, int SourceSize_83,
         int *conflictD_51, int ObjectSize_84,
         int SourceSize_85, int *adjacentListD_52,
         int SourceSize_86, int OffsetEnd_87,
         int *colors_53, const int maxDegree_54) {
*/

   GC_49 << <gridDim.x, blockDim.x>>> 
          (part_boundaryList_37 + (0 + (threadIdx.x * tile_35)), tile_dyn, 
          tile_dyn /*x_size_34*/, OffsetEnd /*(p_33 * tile_35)*/, /*p_33,*/
          part_conflictD_38 + (0 + (threadIdx.x * tile_35)), tile_dyn /*x_size_34*/, 
          OffsetEnd /*SourceSize_103*/, adjacentListD_30, 
          SourceSize_104, OffsetEnd_105, 
          colors_31, maxDegree_32);



   /* GC_49 << <dimGrid, dimBlock>>>
        (part_boundaryList_37, x_size_34, SourceSize_101, (p_33 * tile_35),
         p_33, part_conflictD_38, x_size_34, x_size_34, SourceSize_103,
         adjacentListD_30, SourceSize_104, OffsetEnd_105, colors_31,
         maxDegree_32);
    */
    hipDeviceSynchronize();

  } else {


    GC_39(part_boundaryList_37 + (0 + (threadIdx.x * 1)), x_size_34,
          SourceSize_101, (p_33 * tile_35), p_33,
          part_conflictD_38 + (0 + (threadIdx.x * 1)), x_size_34, x_size_34,
          SourceSize_103, adjacentListD_30, SourceSize_104, OffsetEnd_105,
          colors_31, maxDegree_32);

    __syncthreads();
  }

}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void GC_15(int *boundaryListD_16, int ObjectSize_109, int *conflictD_17,
           int *adjacentListD_18, int SourceSize_110, int OffsetEnd_111,
           int SourceSize_112, int OffsetEnd_113, int *colors_19,
           const int maxDegree_20) {

  int p_21 = TGM_TEMPLATE_0;
  int x_size_22 = ObjectSize_109;
  int tile_23 = ((((x_size_22 + p_21) - 1)) / p_21);

  int *part_boundaryList_25 = boundaryListD_16;
  int *part_conflictD_26 = conflictD_17;

  dim3 dimBlock(TGM_TEMPLATE_1);
  dim3 dimGrid(p_21);
  GC_27 << <dimGrid, dimBlock>>>
      (part_boundaryList_25, tile_23, tile_23, x_size_22, part_conflictD_26,
       tile_23, x_size_22, adjacentListD_18, SourceSize_110, OffsetEnd_111,
       SourceSize_112, OffsetEnd_113, colors_19, maxDegree_20);
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void GC_tangram(
               int *adjacentListD_4, int *boundaryListD_5, int *colors_6,
               int *conflictD_7, long size_8, int boundarySize_9,
               int maxDegree_10) {

  int *Adjacent_List_11;
  hipMalloc((void **)&Adjacent_List_11,
             ((size_8 * maxDegree_10)) * sizeof(int));
  hipMemcpy(Adjacent_List_11, adjacentListD_4,
             ((size_8 * maxDegree_10)) * sizeof(int), hipMemcpyHostToDevice);
  int *Boundary_List_12;
  hipMalloc((void **)&Boundary_List_12, (boundarySize_9) * sizeof(int));
  hipMemcpy(Boundary_List_12, boundaryListD_5, (boundarySize_9) * sizeof(int),
             hipMemcpyHostToDevice);
  int *Colors_13;
  hipMalloc((void **)&Colors_13, (size_8) * sizeof(int));
  hipMemcpy(Colors_13, colors_6, (size_8) * sizeof(int),
             hipMemcpyHostToDevice);
  int *ConflictD_14;
  hipMalloc((void **)&ConflictD_14, (boundarySize_9) * sizeof(int));
  hipMemcpy(ConflictD_14, conflictD_7, (boundarySize_9) * sizeof(int),
             hipMemcpyHostToDevice);

  GC_15<TGM_TEMPLATE_0, TGM_TEMPLATE_1>(
      Boundary_List_12, boundarySize_9, ConflictD_14, Adjacent_List_11,
      (size_8 * maxDegree_10), (size_8 * maxDegree_10), (size_8 * maxDegree_10),
      (size_8 * maxDegree_10), Colors_13, maxDegree_10);

  hipMemcpy(boundaryListD_5, Boundary_List_12, (boundarySize_9) * sizeof(int),
             hipMemcpyDeviceToHost);

  hipFree(Adjacent_List_11);
  hipFree(Boundary_List_12);
  hipFree(Colors_13);
  hipFree(ConflictD_14);
}

void launch_kernel(unsigned int dimGrid_confl, unsigned int dimBlock_confl, int *adjacentListD, int *boundaryListD, int *colors, int *conflictD, long size, int boundarySize, int maxDegree) {
    GC_tangram<2, 32>(adjacentListD, boundaryListD, colors, conflictD, size, boundarySize, maxDegree);
}

