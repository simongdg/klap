#include "hip/hip_runtime.h"
#include "common.h"


__inline__ __device__ void GC_39(int *boundaryListD_40, int SourceSize_49,
                                 int OffsetEnd_50, int ObjectSize_51,
                                 int Stride_52, int *conflictD_41,
                                 int SourceSize_53, int SourceSize_54,
                                 int OffsetEnd_55, int *adjacentListD_42,
                                 int SourceSize_56, int OffsetEnd_57,
                                 int *colors_43, const int maxDegree_44) {

  unsigned int tid_58 = threadIdx.x;
  int i_45;
  for (int idx_46 = 0; (idx_46 < ObjectSize_51); idx_46 += Stride_52) {
    if ((idx_46 + threadIdx.x < SourceSize_49) &&
        (idx_46 + (blockIdx.x * SourceSize_49 + threadIdx.x) < OffsetEnd_50)) {
      //if ((idx_46 + threadIdx.x < SourceSize_54) &&
      //    (idx_46 + (blockIdx.x * SourceSize_54 + threadIdx.x) <
      //     OffsetEnd_55)) {
        i_45 = boundaryListD_40[idx_46];
        conflictD_41[idx_46] = 0;
        for (int k_47 = 0; (k_47 < maxDegree_44); ++k_47) {
          int j_48 = adjacentListD_42[((i_45 * maxDegree_44) + k_47)];
          if (((j_48 < i_45) && ((colors_43[i_45] == colors_43[j_48])))) {
            if (blockIdx.x * blockDim.x + threadIdx.x < SourceSize_53) {
              atomicMax(&conflictD_41[idx_46], (i_45 + 1));
              colors_43[i_45] = 0;
            }
          }
        }
      //} 
    }
  }
}

__global__ void GC_27(int *boundaryListD_28, int ObjectSize_59,
                      int ObjectSize_60, int SourceSize_61, int *conflictD_29,
                      int ObjectSize_62, int SourceSize_63,
                      int *adjacentListD_30, int SourceSize_64,
                      int OffsetEnd_65, int *colors_31,
                      const int maxDegree_32) {

  unsigned int blockID_66 = blockIdx.x;
  int p_33 = blockDim.x;
  int x_size_34 = ObjectSize_60;
  int tile_35 = ((((x_size_34 + p_33) - 1)) / p_33);

  int *part_boundaryList_37 = boundaryListD_28 + (blockIdx.x * ObjectSize_59);
  int *part_conflictD_38 = conflictD_29 + (blockIdx.x * ObjectSize_62);


  GC_39(part_boundaryList_37 + (0 + (threadIdx.x * 1)), x_size_34,
        SourceSize_61, (p_33 * tile_35), p_33,
        part_conflictD_38 + (0 + (threadIdx.x * 1)), x_size_34, x_size_34,
        SourceSize_63, adjacentListD_30, SourceSize_64, OffsetEnd_65, colors_31,
        maxDegree_32);

  __syncthreads();
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void GC_15(int *boundaryListD_16, int ObjectSize_67, int *conflictD_17,
           int *adjacentListD_18, int SourceSize_68, int OffsetEnd_69,
           int *colors_19, const int maxDegree_20) {

  int p_21 = TGM_TEMPLATE_0;
  int x_size_22 = ObjectSize_67;
  int tile_23 = ((((x_size_22 + p_21) - 1)) / p_21);

  int *part_boundaryList_25 = boundaryListD_16;
  int *part_conflictD_26 = conflictD_17;

  dim3 dimBlock(TGM_TEMPLATE_1);
  dim3 dimGrid(p_21);

  GC_27 << <dimGrid, dimBlock>>> (part_boundaryList_25, tile_23, tile_23,
                                  x_size_22, part_conflictD_26, tile_23,
                                  x_size_22, adjacentListD_18, SourceSize_68,
                                  OffsetEnd_69, colors_19, maxDegree_20);
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void GC_tangram(
               int *adjacentListD_4, int *boundaryListD_5, int *colors_6,
               int *conflictD_7, long size_8, int boundarySize_9,
               int maxDegree_10) {

  int *Adjacent_List_11;
  hipMalloc((void **)&Adjacent_List_11,
             ((size_8 * maxDegree_10)) * sizeof(int));
  hipMemcpy(Adjacent_List_11, adjacentListD_4,
             ((size_8 * maxDegree_10)) * sizeof(int), hipMemcpyHostToDevice);
  int *Boundary_List_12;
  hipMalloc((void **)&Boundary_List_12, (boundarySize_9) * sizeof(int));
  hipMemcpy(Boundary_List_12, boundaryListD_5, (boundarySize_9) * sizeof(int),
             hipMemcpyHostToDevice);
  int *Colors_13;
  hipMalloc((void **)&Colors_13, (size_8) * sizeof(int));
  hipMemcpy(Colors_13, colors_6, (size_8) * sizeof(int),
             hipMemcpyHostToDevice);
  int *ConflictD_14;
  hipMalloc((void **)&ConflictD_14, (boundarySize_9) * sizeof(int));
  hipMemcpy(ConflictD_14, conflictD_7, (boundarySize_9) * sizeof(int),
             hipMemcpyHostToDevice);

  GC_15<TGM_TEMPLATE_0, TGM_TEMPLATE_1>(
      Boundary_List_12, boundarySize_9, ConflictD_14, Adjacent_List_11,
      (size_8 * maxDegree_10), (size_8 * maxDegree_10), Colors_13,
      maxDegree_10);

  hipFree(Adjacent_List_11);
  hipFree(Boundary_List_12);
  hipFree(Colors_13);
  hipFree(ConflictD_14);
}

void launch_kernel(unsigned int dimGrid_confl, unsigned int dimBlock_confl, int *adjacentListD, int *boundaryListD, int *colors, int *conflictD, long size, int boundarySize, int maxDegree) {
    GC_tangram<GRID_DIM, BLOCK_DIM>(adjacentListD, boundaryListD, colors, conflictD, size, boundarySize, maxDegree);
}

