#include "hip/hip_runtime.h"
#include "common.h"

__inline__ __device__ void
BFS_74(unsigned int *p_levels_75, int SourceSize_88, int OffsetEnd_89,
       int ObjectSize_90, int Stride_91, unsigned int *levels_76,
       unsigned int *edgeArray_77, unsigned int *edgeArrayAux_78,
       int SourceSize_92, int OffsetEnd_93, const unsigned int numVerts_79,
       int curr_80, int *flag_81) {

  unsigned int tid_94 = threadIdx.x;
  for (int i_83 = 0; (i_83 < ObjectSize_90); i_83 += Stride_91) {
    if ((i_83 + threadIdx.x < SourceSize_88) &&
        (i_83 + (blockIdx.x * SourceSize_88 + threadIdx.x) < OffsetEnd_89)) {
      if ((p_levels_75[i_83] == curr_80)) {
        unsigned int nbr_off_84 = edgeArray_77[i_83];
        unsigned int num_nbr_85 = (edgeArray_77[(i_83 + 1)] - nbr_off_84);
        for (int nbr_count_86 = 0; (nbr_count_86 < num_nbr_85);
             ++nbr_count_86) {
          int v_87 = edgeArrayAux_78[(nbr_count_86 + nbr_off_84)];
          if ((levels_76[v_87] == UINT_MAX)) {
            levels_76[v_87] = (curr_80 + 1);
            flag_81[0] = 1;
          }
        }
      }
    }
  }
}

__global__ void
BFS_58(unsigned int *p_levels_59, int ObjectSize_95, int SourceSize_96,
       unsigned int *levels_60, int ObjectSize_97, unsigned int *edgeArray_61,
       int ObjectSize_98, unsigned int *edgeArrayAux_62, int SourceSize_99,
       int OffsetEnd_100, const unsigned int numVerts_63, int curr_64,
       int *flag_65) {

  unsigned int tid_101 = blockIdx.x;
  int p_68 = blockDim.x;
  int x_size_69 = ObjectSize_97;
  int tile_70 = ((((x_size_69 + p_68) - 1)) / p_68);

  unsigned int *part_levels_72 = p_levels_59 + (blockIdx.x * ObjectSize_95);
  unsigned int *part_edgeArray_73 = edgeArray_61 + (blockIdx.x * ObjectSize_98);


  BFS_74(part_levels_72 + (0 + (threadIdx.x * 1)), x_size_69, SourceSize_96,
         (p_68 * tile_70), p_68, levels_60,
         part_edgeArray_73 + (0 + (threadIdx.x * 1)), edgeArrayAux_62,
         SourceSize_99, OffsetEnd_100, numVerts_63, curr_64, flag_65);

  __syncthreads();
}

__inline__ __device__ void
BFS_44(unsigned int *p_levels_45, int SourceSize_102, int OffsetEnd_103,
       int ObjectSize_104, int Stride_105, unsigned int *levels_46,
       unsigned int *edgeArray_47, unsigned int *edgeArrayAux_48,
       int SourceSize_106, int OffsetEnd_107, const unsigned int numVerts_49,
       int curr_50, int *flag_51) {

  unsigned int tid_108 = threadIdx.x;
  for (int i_53 = 0; (i_53 < ObjectSize_104); i_53 += Stride_105) {
    if ((i_53 + threadIdx.x < SourceSize_102) &&
        (i_53 + (blockIdx.x * SourceSize_102 + threadIdx.x) < OffsetEnd_103)) {
      //      p_levels_45[i_53] = curr_50;
      if ((p_levels_45[i_53] == curr_50)) {
        unsigned int nbr_off_54 = edgeArray_47[i_53];
        unsigned int num_nbr_55 = (edgeArray_47[(i_53 + 1)] - nbr_off_54);
        for (int nbr_count_56 = 0; (nbr_count_56 < num_nbr_55);
             ++nbr_count_56) {
          int v_57 = edgeArrayAux_48[(nbr_count_56 + nbr_off_54)];
          if ((levels_46[v_57] == UINT_MAX)) {
            levels_46[v_57] = (curr_50 + 1);
            flag_51[0] = 1;
          }
        }
      }
    }
  }
}

__global__ void BFS_28(unsigned int *p_levels_29, int ObjectSize_109,
                       int SourceSize_110, unsigned int *levels_30,
                       int ObjectSize_111, int ObjectSize_112,
                       unsigned int *edgeArray_31, int ObjectSize_113,
                       unsigned int *edgeArrayAux_32, int SourceSize_114,
                       int OffsetEnd_115, int SourceSize_116, int OffsetEnd_117,
                       const unsigned int numVerts_33, int curr_34,
                       int *flag_35) {

  unsigned int blockID_118 = blockIdx.x;
  /*Vector*/;
  unsigned int num_nbr_37 =
      (edgeArray_31[(blockIdx.x * blockDim.x + threadIdx.x + 1)] -
       edgeArray_31[blockIdx.x * blockDim.x + threadIdx.x]);
  int p_38 = blockDim.x;
  int x_size_39 = ObjectSize_111;
  int tile_40 = ((((x_size_39 + p_38) - 1)) / p_38);

  unsigned int *part_levels_42 = p_levels_29 + (blockIdx.x * ObjectSize_109);
  unsigned int *part_edgeArray_43 =
      edgeArray_31 + (blockIdx.x * ObjectSize_113);
  /*DYNAMIC*/
  if (((tile_40 > 1) && (num_nbr_37 > 10))) {
    //printf("Dyamic \n");
    int tile_dyn = ((((tile_40 + gridDim.x) - 1)) / gridDim.x);

    int blockBound = (blockIdx.x < (gridDim.x - 1))
                         ? x_size_39
                         : (SourceSize_110 - (blockIdx.x * x_size_39));

    int OffsetEnd = (threadIdx.x < (blockDim.x - 1))
                        ? tile_40
                        : (blockBound - (threadIdx.x * tile_40));

    BFS_58 << <gridDim.x, blockDim.x>>>
        (part_levels_42 + (threadIdx.x*tile_40), tile_dyn /*x_size_39*/,
         OffsetEnd /*(p_38 * tile_40)*/, /*p_38,*/
         levels_30, tile_dyn, part_edgeArray_43 + (threadIdx.x*tile_40), tile_dyn, edgeArrayAux_32, SourceSize_114,
         OffsetEnd_115, numVerts_33, curr_34, flag_35);

    hipDeviceSynchronize(); // could remove

  } else {

    BFS_44(part_levels_42 + (0 + (threadIdx.x * 1)), x_size_39, SourceSize_110,
           (p_38 * tile_40), p_38, levels_30,
           part_edgeArray_43 + (0 + (threadIdx.x * 1)), edgeArrayAux_32,
           SourceSize_114, OffsetEnd_115, numVerts_33, curr_34, flag_35);

    __syncthreads(); // could remove
  }
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void BFS_14(unsigned int *p_levels_15, unsigned int *levels_16,
            int ObjectSize_119, int ObjectSize_120, int ObjectSize_121,
            unsigned int *edgeArray_17, unsigned int *edgeArrayAux_18,
            int SourceSize_122, int OffsetEnd_123, int SourceSize_124,
            int OffsetEnd_125, const unsigned int numVerts_19, int curr_20,
            int *flag_21) {

  int p_22 = TGM_TEMPLATE_0;
  int x_size_23 = ObjectSize_119;
  int tile_24 = ((((x_size_23 + p_22) - 1)) / p_22);

  unsigned int *part_levels_26 = p_levels_15;
  unsigned int *part_edgeArray_27 = edgeArray_17;

  dim3 dimBlock(TGM_TEMPLATE_1/*(((x_size_23 - 1) / p_22) + 1)*/);
  dim3 dimGrid(p_22);

  BFS_28 << <dimGrid, dimBlock>>>
      (part_levels_26, tile_24, x_size_23, levels_16, ObjectSize_120,
       ObjectSize_121, part_edgeArray_27, tile_24, edgeArrayAux_18,
       SourceSize_122, OffsetEnd_123, SourceSize_124, OffsetEnd_125,
       numVerts_19, curr_20, flag_21);
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void BFS_tangram(unsigned int *d_costArray_2, unsigned int *d_edgeArray_3,
                unsigned int *d_edgeArrayAux_4, unsigned int numVerts_5,
                unsigned int adj_list_length_6, int iters_7, int *flag_8) {

  BFS_14<TGM_TEMPLATE_0, TGM_TEMPLATE_1>(
      d_costArray_2, d_costArray_2, numVerts_5, numVerts_5, numVerts_5,
      d_edgeArray_3, d_edgeArrayAux_4, adj_list_length_6, adj_list_length_6,
      adj_list_length_6, adj_list_length_6, numVerts_5, iters_7, flag_8);

}

void launch_kernel(unsigned int *d_costArray, unsigned int *d_edgeArray,
                   unsigned int *d_edgeArrayAux, unsigned int numVerts,
                   int iters, int *d_flag) {
  //unsigned int numBlocks = (numVerts - 1) / PARENT_BLOCK_SIZE + 1;
  BFS_tangram<GRID_DIM, BLOCK_DIM>(d_costArray, d_edgeArray, d_edgeArrayAux, numVerts, 1,
                      iters, d_flag);
}
