#include "hip/hip_runtime.h"
#include "common.h"

__inline__ __device__ void
BFS_42(unsigned int *p_levels_43, int SourceSize_56, int OffsetEnd_57,
       int ObjectSize_58, int Stride_59, unsigned int *levels_44,
       unsigned int *edgeArray_45, unsigned int *edgeArrayAux_46,
       int SourceSize_60, int OffsetEnd_61, const unsigned int numVerts_47,
       int curr_48, int *flag_49) {

  unsigned int tid_62 = threadIdx.x;
  for (int i_51 = 0; (i_51 < ObjectSize_58); i_51 += Stride_59) {
    if ((i_51 + threadIdx.x < SourceSize_56) &&
        (i_51 + (blockIdx.x * SourceSize_56 + threadIdx.x) < OffsetEnd_57)) {
      if ((p_levels_43[i_51] == curr_48)) {
        unsigned int nbr_off_52 = edgeArray_45[i_51];
        unsigned int num_nbr_53 = (edgeArray_45[(i_51 + 1)] - nbr_off_52);
        for (int nbr_count_54 = 0; (nbr_count_54 < num_nbr_53);
             ++nbr_count_54) {
          int v_55 = edgeArrayAux_46[(nbr_count_54 + nbr_off_52)];
          if ((levels_44[v_55] == UINT_MAX)) {
            levels_44[v_55] = (curr_48 + 1);
            flag_49[0] = 1;
          }
        }
      }
    }
  }
}

__global__ void BFS_28(unsigned int *p_levels_29, int ObjectSize_63,
                       int SourceSize_64, unsigned int *levels_30,
                       int ObjectSize_65, unsigned int *edgeArray_31,
                       int ObjectSize_66, unsigned int *edgeArrayAux_32,
                       int SourceSize_67, int OffsetEnd_68,
                       const unsigned int numVerts_33, int curr_34,
                       int *flag_35) {

  unsigned int blockID_69 = blockIdx.x;
  int p_36 = blockDim.x;
  int x_size_37 = ObjectSize_65;
  int tile_38 = ((((x_size_37 + p_36) - 1)) / p_36);

  unsigned int *part_levels_40 = p_levels_29 + (blockIdx.x * ObjectSize_63);
  unsigned int *part_edgeArray_41 = edgeArray_31 + (blockIdx.x * ObjectSize_66);

//  __shared__ void *map_return_4;
//  if (threadIdx.x == 0) {
//    map_return_4 = new void[p_36];
//  }

//  __syncthreads();

  BFS_42(part_levels_40 + (0 + (threadIdx.x * 1)), x_size_37, SourceSize_64,
         (p_36 * tile_38), p_36, levels_30,
         part_edgeArray_41 + (0 + (threadIdx.x * 1)), edgeArrayAux_32,
         SourceSize_67, OffsetEnd_68, numVerts_33, curr_34, flag_35);

  __syncthreads();
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void BFS_14(unsigned int *p_levels_15, unsigned int *levels_16,
            int ObjectSize_70, int ObjectSize_71, unsigned int *edgeArray_17,
            unsigned int *edgeArrayAux_18, int SourceSize_72, int OffsetEnd_73,
            const unsigned int numVerts_19, int curr_20, int *flag_21) {

  int p_22 = TGM_TEMPLATE_0;
  int x_size_23 = ObjectSize_70;
  int tile_24 = ((((x_size_23 + p_22) - 1)) / p_22);

  unsigned int *part_levels_26 = p_levels_15;
  unsigned int *part_edgeArray_27 = edgeArray_17;

//  void *map_return_h_2 = new void[p_22];
//  void *map_return_1;
//  hipMalloc((void **)&map_return_1, (p_22) * sizeof(void));
  dim3 dimBlock(TGM_TEMPLATE_1);
  dim3 dimGrid(p_22);
  BFS_28 << <dimGrid, dimBlock>>> (part_levels_26, tile_24, x_size_23,
                                   levels_16, ObjectSize_71, part_edgeArray_27,
                                   tile_24, edgeArrayAux_18, SourceSize_72,
                                   OffsetEnd_73, numVerts_19, curr_20, flag_21);

//  hipMemcpy(map_return_h_2, map_return_1, (p_22) * sizeof(void),
//             hipMemcpyDeviceToHost);
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void BFS_tangram(unsigned int *d_costArray_2, unsigned int *d_edgeArray_3,
                unsigned int *d_edgeArrayAux_4, unsigned int numVerts_5,
                unsigned int adj_list_length_6, int iters_7, int *flag_8) {
/*  unsigned int *COST_ARRAY_P_9;
  hipMalloc((void **)&COST_ARRAY_P_9, (numVerts_5) * sizeof(unsigned int));
  hipMemcpy(COST_ARRAY_P_9, h_costArray_2, (numVerts_5) * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  unsigned int *COST_ARRAY_10;
  hipMalloc((void **)&COST_ARRAY_10, (numVerts_5) * sizeof(unsigned int));
  hipMemcpy(COST_ARRAY_10, h_costArray_2, (numVerts_5) * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  unsigned int *EDGE_ARRAY_11;
  hipMalloc((void **)&EDGE_ARRAY_11,
             ((numVerts_5 + 1)) * sizeof(unsigned int));
  hipMemcpy(EDGE_ARRAY_11, h_edgeArray_3,
             ((numVerts_5 + 1)) * sizeof(unsigned int), hipMemcpyHostToDevice);
  unsigned int *EDGE_ARRAY_AUX_12;
  hipMalloc((void **)&EDGE_ARRAY_AUX_12,
             (adj_list_length_6) * sizeof(unsigned int));
  hipMemcpy(EDGE_ARRAY_AUX_12, h_edgeArrayAux_4,
             (adj_list_length_6) * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  int *FLAG_13;
  hipMalloc((void **)&FLAG_13, (1) * sizeof(int));
  hipMemcpy(FLAG_13, flag_8, (1) * sizeof(int), hipMemcpyHostToDevice);
*/
  BFS_14<TGM_TEMPLATE_0, TGM_TEMPLATE_1>(
      d_costArray_2, d_costArray_2, numVerts_5, numVerts_5, d_edgeArray_3,
      d_edgeArrayAux_4, adj_list_length_6, adj_list_length_6, numVerts_5,
      iters_7, flag_8);

//  hipMemcpy(h_costArray_2, COST_ARRAY_P_9, (numVerts_5) * sizeof(unsigned int),
//             hipMemcpyDeviceToHost);

//  hipFree(COST_ARRAY_P_9);
//  hipFree(COST_ARRAY_10);
//  hipFree(EDGE_ARRAY_11);
//  hipFree(EDGE_ARRAY_AUX_12);
//  hipFree(FLAG_13);
}

void launch_kernel(unsigned int *d_costArray, unsigned int *d_edgeArray,
                   unsigned int *d_edgeArrayAux, unsigned int numVerts,
                   int iters, int *d_flag) {
  unsigned int numBlocks = (numVerts - 1) / PARENT_BLOCK_SIZE + 1;
  //printf("numBlocks = %d\n", numBlocks);
  //std::cout<<"numBlocks = " << numBlocks <<", PARENT_BLOCK_SIZE = " << PARENT_BLOCK_SIZE + 1 <<std::endl;
  BFS_tangram<GRID_DIM, BLOCK_DIM>(d_costArray, d_edgeArray, d_edgeArrayAux, numVerts, 1,
                      iters, d_flag);
}
