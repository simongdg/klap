#include "hip/hip_runtime.h"
#include "common.h"

__inline__ __device__ void
MST_71(unsigned int *outgoing_72, int SourceSize_94, int OffsetEnd_95,
       int ObjectSize_96, int Stride_97, unsigned int *ele2comp_p_73,
       int SourceSize_98, int OffsetEnd_99, unsigned int *ele2comp_74,
       unsigned int *eleminwts_75, unsigned int *minwtcomponent_76,
       unsigned int *partners_77, unsigned int *srcsrc_78,
       unsigned int *psrc_79, unsigned int *edgessrcdst_80,
       unsigned int *edgessrcwt_81, unsigned int *goaheadnodeofcomponent_82,
       unsigned int numNodes_83, unsigned int numEdges_84) {

  unsigned int tid_100 = threadIdx.x;
  for (int nn_85 = 0; (nn_85 < ObjectSize_96); nn_85 += Stride_97) {
    if ((nn_85 + threadIdx.x < SourceSize_98) &&
        (nn_85 + (blockIdx.x * SourceSize_98 + threadIdx.x) < OffsetEnd_99)) {
      unsigned int element_87 = (nn_85 + (blockIdx.x * SourceSize_98 + threadIdx.x)) ;
      while ((((atomicCAS(&ele2comp_74[element_87],element_87, element_87)) ==
              element_87)) == false) {
        element_87 = ele2comp_74[element_87];
      }
      ele2comp_p_73[nn_85] = element_87;
      if (((((eleminwts_75[nn_85] == minwtcomponent_76[element_87])) &&
            ((element_87 != partners_77[nn_85]))) &&
           ((partners_77[nn_85] != numNodes_83)))) {
         unsigned int degree_88 = outgoing_72[nn_85];
        for (int ii_89 = 0; (ii_89 < degree_88); ++ii_89) {
          if ((srcsrc_78[nn_85] < numNodes_83)) {
            unsigned int edge_90 = (psrc_79[srcsrc_78[nn_85]] + ii_89);
            if ((edge_90 && (edge_90 < (numEdges_84 + 1)))) {
              unsigned int wt_91 = edgessrcwt_81[edge_90];
              if ((wt_91 == eleminwts_75[nn_85])) {
                unsigned int dst_92 = edgessrcdst_80[edge_90];
                unsigned int dst_element_93 = dst_92;
                while ((((atomicCAS(&ele2comp_74[dst_element_93], dst_element_93,
                          dst_element_93)) == dst_element_93)) == false) {
                  dst_element_93 = ele2comp_74[dst_element_93];
                }
                ele2comp_74[dst_92] = dst_element_93;
                if ((dst_element_93 == partners_77[nn_85])) {
                  if (((atomicCAS(&goaheadnodeofcomponent_82[element_87], numNodes_83,
                        (nn_85 + (blockIdx.x * SourceSize_98 + threadIdx.x)) )) == numNodes_83)) {
                        
                  }
                }
              }
            }
          }
        }
      }
    }
  }
}

__global__ void
MST_48(unsigned int *outgoing_49, int ObjectSize_101, int ObjectSize_102,
       int SourceSize_103, unsigned int *ele2comp_p_50, int ObjectSize_104,
       int SourceSize_105, unsigned int *ele2comp_51,
       unsigned int *eleminwts_52, int ObjectSize_106,
       unsigned int *minwtcomponent_53, unsigned int *partners_54,
       int ObjectSize_107, unsigned int *srcsrc_55, int ObjectSize_108,
       unsigned int *psrc_56, unsigned int *edgessrcdst_57,
       unsigned int *edgessrcwt_58, unsigned int *goaheadnodeofcomponent_59,
       unsigned int numNodes_60, unsigned int numEdges_61) {

  unsigned int blockID_109 = blockIdx.x;
  int p_62 = blockDim.x;
  int x_size_63 = ObjectSize_102;
  int tile_64 = ((((x_size_63 + p_62) - 1)) / p_62);

  unsigned int *part_outgoing_66 = outgoing_49 + (blockIdx.x * ObjectSize_101);
  unsigned int *part_ele2comp_67 =
      ele2comp_p_50 + (blockIdx.x * ObjectSize_104);
  unsigned int *part_eleminwts_68 =
      eleminwts_52 + (blockIdx.x * ObjectSize_106);
  unsigned int *part_partners_69 = partners_54 + (blockIdx.x * ObjectSize_107);
  unsigned int *part_srcsrc_70 = srcsrc_55 + (blockIdx.x * ObjectSize_108);

  MST_71(part_outgoing_66 + (0 + (threadIdx.x * 1)), x_size_63, SourceSize_103,
         (p_62 * tile_64), p_62, part_ele2comp_67 + (0 + (threadIdx.x * 1)),
         x_size_63, SourceSize_105, ele2comp_51,
         part_eleminwts_68 + (0 + (threadIdx.x * 1)), minwtcomponent_53,
         part_partners_69 + (0 + (threadIdx.x * 1)),
         part_srcsrc_70 + (0 + (threadIdx.x * 1)), psrc_56, edgessrcdst_57,
         edgessrcwt_58, goaheadnodeofcomponent_59, numNodes_60, numEdges_61);

  __syncthreads();
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void MST_25(unsigned int *outgoing_26, int ObjectSize_110,
            unsigned int *ele2comp_p_27, unsigned int *ele2comp_28,
            unsigned int *eleminwts_29, unsigned int *minwtcomponent_30,
            unsigned int *partners_31, unsigned int *srcsrc_32,
            unsigned int *psrc_33, unsigned int *edgessrcdst_34,
            unsigned int *edgessrcwt_35,
            unsigned int *goaheadnodeofcomponent_36, unsigned int numNodes_37,
            unsigned int numEdges_38) {

  int p_39 = TGM_TEMPLATE_0;
  int x_size_40 = ObjectSize_110;
  int tile_41 = ((((x_size_40 + p_39) - 1)) / p_39);

  unsigned int *part_outgoing_43 = outgoing_26;
  unsigned int *part_ele2comp_44 = ele2comp_p_27;
  unsigned int *part_eleminwts_45 = eleminwts_29;
  unsigned int *part_partners_46 = partners_31;
  unsigned int *part_srcsrc_47 = srcsrc_32;

  dim3 dimBlock(TGM_TEMPLATE_1);
  dim3 dimGrid(p_39);

  MST_48 << <dimGrid, dimBlock>>>
      (part_outgoing_43, tile_41, tile_41, x_size_40, part_ele2comp_44, tile_41,
       x_size_40, ele2comp_28, part_eleminwts_45, tile_41, minwtcomponent_30,
       part_partners_46, tile_41, part_srcsrc_47, tile_41, psrc_33,
       edgessrcdst_34, edgessrcwt_35, goaheadnodeofcomponent_36, numNodes_37,
       numEdges_38);
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void MST_tangram(unsigned int *ele2comp_2, unsigned int *eleminwts_3,
                 unsigned int *minwtcomponen_4, unsigned int *partners_5,
                 unsigned int *outgoing_6, unsigned int *srcsrc_7,
                 unsigned int *psrc_8, unsigned int *edgessrcdst_9,
                 unsigned int *edgessrcwt_10,
                 unsigned int *goaheadnodeofcomponent_11,
                 unsigned int numNodes_12, unsigned int numEdges_13) {

  unsigned int *T_ele2comp_14 = ele2comp_2;
  unsigned int *T_ele2comp_p_15 = ele2comp_2;
  unsigned int *T_eleminwts_16 = eleminwts_3;
  unsigned int *T_minwtcomponen_17 = minwtcomponen_4;
  unsigned int *T_partners_18 = partners_5;
  unsigned int *T_outgoing_19 = outgoing_6;
  unsigned int *T_srcsrc_20 = srcsrc_7;
  unsigned int *T_psrc_21 = psrc_8;
  unsigned int *T_edgessrcdst_22 = edgessrcdst_9;
  unsigned int *T_edgessrcwt_23 = edgessrcwt_10;
  unsigned int *T_goaheadnodeofcomponent_24 = goaheadnodeofcomponent_11;

  MST_25<TGM_TEMPLATE_0, TGM_TEMPLATE_1>(
      T_outgoing_19, numNodes_12, T_ele2comp_p_15, T_ele2comp_14,
      T_eleminwts_16, T_minwtcomponen_17, T_partners_18, T_srcsrc_20, T_psrc_21,
      T_edgessrcdst_22, T_edgessrcwt_23, T_goaheadnodeofcomponent_24,
      numNodes_12, numEdges_13);
}

void launch_find_kernel(unsigned int nb, unsigned int nt, unsigned *mstwt,
                        Graph graph, ComponentSpace cs, foru *eleminwts,
                        foru *minwtcomponent, unsigned *partners,
                        unsigned *phore, bool *processinnextiteration,
                        unsigned *goaheadnodeofcomponent, unsigned inpid) {

  MST_tangram<4, 1024>(cs.getEle2comp(), (unsigned int *)eleminwts,
                       (unsigned int *)minwtcomponent, partners,
                       graph.getNoutGoing(), graph.getSrcsrc(), graph.getPsrc(),
                       graph.getEdgessrcdst(), graph.getEdgessrcwt(),
                       goaheadnodeofcomponent, graph.getNnodes(),
                       graph.getNedges());
}


////////////////////////////////////////////////////////////////////////////////////


__global__ void verify_min_elem_child_cdp(Graph graph, ComponentSpace cs, unsigned minwt_node, foru minwt, /*foru *eleminwts,*/ unsigned *partners, bool *processinnextiteration, /*unsigned *goaheadnodeofcomponent, unsigned src,*/ unsigned id, /*unsigned srcboss,*/ unsigned degree) {
    //bool minwt_found = false;
    unsigned ii = blockIdx.x * blockDim.x + threadIdx.x;
    if (ii < degree){
        foru wt = graph.getWeight(minwt_node, ii);
        //printf("%d: looking at %d edge %d wt %d (%d)\n", id, minwt_node, ii, wt, minwt);

        if (wt == minwt) {
            //minwt_found = true;
            unsigned dst = graph.getDestination(minwt_node, ii);
            unsigned tempdstboss = cs.find(dst);
            if(tempdstboss == partners[minwt_node] && tempdstboss != id)
            {
                processinnextiteration[minwt_node] = true;
                //printf("%d okay!\n", id);
                return;
            }
        }
    }
    else return;
    //printf("component %d is wrong %d - %d - %d, %d\n", id, minwt_found, minwt, ii, degree); // Thread that would printf "okay" should set a verify[id] element. After child kernel finish, if verify[id]!=1, then printf this wrong message
}

__global__ void verify_min_elem_cdp(unsigned *mstwt, Graph graph, ComponentSpace cs, foru *eleminwts, foru *minwtcomponent, unsigned *partners, unsigned *phore, bool *processinnextiteration, unsigned *goaheadnodeofcomponent, unsigned inpid) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if (inpid < graph.nnodes) id = inpid;


    if (id < graph.nnodes) {

        if(cs.isBoss(id)) {

            if(goaheadnodeofcomponent[id] != graph.nnodes) {
                unsigned minwt_node = goaheadnodeofcomponent[id];
                unsigned degree = graph.getOutDegree(minwt_node);
                foru minwt = minwtcomponent[id];

                if(minwt != MYINFINITY) {
                    verify_min_elem_child_cdp<<<(int)ceil((float)degree/BLOCK_DIM), BLOCK_DIM>>>(graph, cs, minwt_node, minwt, partners, processinnextiteration, /*goaheadnodeofcomponent, src,*/ id, /*srcboss,*/ degree);
                }
            }
        }
    }

}

void launch_verify_kernel(unsigned int nb, unsigned int nt, unsigned *mstwt, Graph graph, ComponentSpace cs, foru *eleminwts, foru *minwtcomponent, unsigned *partners, unsigned *phore, bool *processinnextiteration, unsigned *goaheadnodeofcomponent, unsigned inpid) {
    verify_min_elem_cdp<<<nb, nt>>>(mstwt, graph, cs, eleminwts, minwtcomponent, partners, phore, processinnextiteration, goaheadnodeofcomponent, inpid);
}


