#include "hip/hip_runtime.h"
#include "common.h"

__inline__ __device__ void
MST_117(unsigned int *outgoing_118, int SourceSize_140, int OffsetEnd_141, int threadOffset,
        int ObjectSize_142, int Stride_143, unsigned int *ele2comp_p_119,
        int SourceSize_144, int OffsetEnd_145, unsigned int *ele2comp_120,
        unsigned int *eleminwts_121, unsigned int *minwtcomponent_122,
        unsigned int *partners_123, unsigned int *srcsrc_124,
        unsigned int *psrc_125, unsigned int *edgessrcdst_126,
        unsigned int *edgessrcwt_127, unsigned int *goaheadnodeofcomponent_128,
        unsigned int numNodes_129, unsigned int numEdges_130) {

  unsigned int tid_146 = threadIdx.x;
  for (int nn_131 = 0; (nn_131 < ObjectSize_142); nn_131 += Stride_143) {
    if ((nn_131 + threadIdx.x < SourceSize_144) &&
        (nn_131 + (blockIdx.x * SourceSize_144 + threadIdx.x) <
         OffsetEnd_145)) {
      unsigned int element_133 = threadOffset +
          (nn_131 + (blockIdx.x * SourceSize_144 + threadIdx.x));
      while ((((atomicCAS(&ele2comp_120[element_133], element_133,
                          element_133)) == element_133)) == false) {
        element_133 = ele2comp_120[element_133];
      }
      ele2comp_p_119[nn_131] = element_133;
      if (((((eleminwts_121[nn_131] == minwtcomponent_122[element_133])) &&
            ((element_133 != partners_123[nn_131]))) &&
           ((partners_123[nn_131] != numNodes_129)))) {
        unsigned int degree_134 = outgoing_118[nn_131];
        for (int ii_135 = 0; (ii_135 < degree_134); ++ii_135) {
          if ((srcsrc_124[nn_131] < numNodes_129)) {
            unsigned int edge_136 = (psrc_125[srcsrc_124[nn_131]] + ii_135);
            if ((edge_136 && (edge_136 < (numEdges_130 + 1)))) {
              unsigned int wt_137 = edgessrcwt_127[edge_136];
              if ((wt_137 == eleminwts_121[nn_131])) {
                unsigned int dst_138 = edgessrcdst_126[edge_136];
                unsigned int dst_element_139 = dst_138;
                while ((((atomicCAS(&ele2comp_120[dst_element_139],
                                    dst_element_139, dst_element_139)) ==
                         dst_element_139)) == false) {
                  dst_element_139 = ele2comp_120[dst_138];
                }
                ele2comp_120[dst_138] = dst_element_139;
                if ((dst_element_139 == partners_123[nn_131])) {
                  if (((atomicCAS(&goaheadnodeofcomponent_128[element_133],
                                  numNodes_129, threadOffset +
                                  (nn_131 + (blockIdx.x * SourceSize_144 +
                                             threadIdx.x)))) == numNodes_129)) {
                  }
                }
              }
            }
          }
        }
      }
    }
  }
}

__global__ void
MST_94(unsigned int *outgoing_95, int ObjectSize_147, int ObjectSize_148, int threadOffset,
       int SourceSize_149, unsigned int *ele2comp_p_96, int ObjectSize_150,
       int SourceSize_151, unsigned int *ele2comp_97,
       unsigned int *eleminwts_98, int ObjectSize_152,
       unsigned int *minwtcomponent_99, unsigned int *partners_100,
       int ObjectSize_153, unsigned int *srcsrc_101, int ObjectSize_154,
       unsigned int *psrc_102, unsigned int *edgessrcdst_103,
       unsigned int *edgessrcwt_104, unsigned int *goaheadnodeofcomponent_105,
       unsigned int numNodes_106, unsigned int numEdges_107) {

  unsigned int tid_155 = blockIdx.x;
  int p_108 = blockDim.x;
  int x_size_109 = ObjectSize_148;
  int tile_110 = ((((x_size_109 + p_108) - 1)) / p_108);

  unsigned int *part_outgoing_112 = outgoing_95 + (blockIdx.x * ObjectSize_147);
  unsigned int *part_ele2comp_113 =
      ele2comp_p_96 + (blockIdx.x * ObjectSize_150);
  unsigned int *part_eleminwts_114 =
      eleminwts_98 + (blockIdx.x * ObjectSize_152);
  unsigned int *part_partners_115 =
      partners_100 + (blockIdx.x * ObjectSize_153);
  unsigned int *part_srcsrc_116 = srcsrc_101 + (blockIdx.x * ObjectSize_154);

  MST_117(
      part_outgoing_112 + (0 + (threadIdx.x * 1)), x_size_109, SourceSize_149, threadOffset,
      (p_108 * tile_110), p_108, part_ele2comp_113 + (0 + (threadIdx.x * 1)),
      x_size_109, SourceSize_151, ele2comp_97,
      part_eleminwts_114 + (0 + (threadIdx.x * 1)), minwtcomponent_99,
      part_partners_115 + (0 + (threadIdx.x * 1)),
      part_srcsrc_116 + (0 + (threadIdx.x * 1)), psrc_102, edgessrcdst_103,
      edgessrcwt_104, goaheadnodeofcomponent_105, numNodes_106, numEdges_107);

  __syncthreads();
}

__inline__ __device__ void
MST_71(unsigned int *outgoing_72, int SourceSize_156, int OffsetEnd_157,
       int ObjectSize_158, int Stride_159, unsigned int *ele2comp_p_73,
       int SourceSize_160, int OffsetEnd_161, unsigned int *ele2comp_74,
       unsigned int *eleminwts_75, unsigned int *minwtcomponent_76,
       unsigned int *partners_77, unsigned int *srcsrc_78,
       unsigned int *psrc_79, unsigned int *edgessrcdst_80,
       unsigned int *edgessrcwt_81, unsigned int *goaheadnodeofcomponent_82,
       unsigned int numNodes_83, unsigned int numEdges_84) {

  unsigned int tid_162 = threadIdx.x;
  for (int nn_85 = 0; (nn_85 < ObjectSize_158); nn_85 += Stride_159) {
    if ((nn_85 + threadIdx.x < SourceSize_160) &&
        (nn_85 + (blockIdx.x * SourceSize_160 + threadIdx.x) < OffsetEnd_161)) {
      unsigned int element_87 =
          (nn_85 + (blockIdx.x * SourceSize_160 + threadIdx.x));
      while ((((atomicCAS(&ele2comp_74[element_87], element_87, element_87)) ==
               element_87)) == false) {
        element_87 = ele2comp_74[element_87];
      }
      ele2comp_p_73[nn_85] = element_87;
      if (((((eleminwts_75[nn_85] == minwtcomponent_76[element_87])) &&
            ((element_87 != partners_77[nn_85]))) &&
           ((partners_77[nn_85] != numNodes_83)))) {
        unsigned int degree_88 = outgoing_72[nn_85];
        for (int ii_89 = 0; (ii_89 < degree_88); ++ii_89) {
          if ((srcsrc_78[nn_85] < numNodes_83)) {
            unsigned int edge_90 = (psrc_79[srcsrc_78[nn_85]] + ii_89);
            if ((edge_90 && (edge_90 < (numEdges_84 + 1)))) {
              unsigned int wt_91 = edgessrcwt_81[edge_90];
              if ((wt_91 == eleminwts_75[nn_85])) {
                unsigned int dst_92 = edgessrcdst_80[edge_90];
                unsigned int dst_element_93 = dst_92;
                while ((((atomicCAS(&ele2comp_74[dst_element_93],
                                    dst_element_93, dst_element_93)) ==
                         dst_element_93)) == false) {
                  dst_element_93 = ele2comp_74[dst_92];
                }
                ele2comp_74[dst_92] = dst_element_93;
                if ((dst_element_93 == partners_77[nn_85])) {
                  if (((atomicCAS(&goaheadnodeofcomponent_82[element_87],
                                  numNodes_83,
                                  (nn_85 + (blockIdx.x * SourceSize_160 +
                                            threadIdx.x)))) == numNodes_83)) {
                  }
                }
              }
            }
          }
        }
      }
    }
  }
}

__global__ void
MST_48(unsigned int *outgoing_49, int ObjectSize_163, int ObjectSize_164,
       int SourceSize_165, unsigned int *ele2comp_p_50, int ObjectSize_166,
       int SourceSize_167, unsigned int *ele2comp_51,
       unsigned int *eleminwts_52, int ObjectSize_168,
       unsigned int *minwtcomponent_53, unsigned int *partners_54,
       int ObjectSize_169, unsigned int *srcsrc_55, int ObjectSize_170,
       unsigned int *psrc_56, unsigned int *edgessrcdst_57,
       unsigned int *edgessrcwt_58, unsigned int *goaheadnodeofcomponent_59,
       unsigned int numNodes_60, unsigned int numEdges_61) {

  unsigned int blockID_171 = blockIdx.x;
  int p_62 = blockDim.x;
  int x_size_63 = ObjectSize_164;
  int tile_64 = ((((x_size_63 + p_62) - 1)) / p_62);

  unsigned int *part_outgoing_66 = outgoing_49 + (blockIdx.x * ObjectSize_163);
  unsigned int *part_ele2comp_67 =
      ele2comp_p_50 + (blockIdx.x * ObjectSize_166);
  unsigned int *part_eleminwts_68 =
      eleminwts_52 + (blockIdx.x * ObjectSize_168);
  unsigned int *part_partners_69 = partners_54 + (blockIdx.x * ObjectSize_169);
  unsigned int *part_srcsrc_70 = srcsrc_55 + (blockIdx.x * ObjectSize_170);
  // if(blockIdx.x == 0 && threadIdx.x == 0)
  //  printf("tile = %d \n", tile_64);
  /*DYNAMIC*/
  if ((tile_64 > 1)) {

    int tile_dyn = ((((tile_64 + gridDim.x) - 1)) / gridDim.x);

    int blockBound = (blockIdx.x < (gridDim.x - 1))
                         ? x_size_63
                         : (SourceSize_165 - (blockIdx.x * x_size_63));

    int OffsetEnd = (threadIdx.x < (blockDim.x - 1))
                        ? tile_64
                        : (blockBound - (threadIdx.x * tile_64));

    int threadOffset = (blockIdx.x * x_size_63) + (threadIdx.x * tile_64);

    /*   MST_94(unsigned int *outgoing_95, int ObjectSize_147,
                int ObjectSize_148, int SourceSize_149,
                unsigned int *ele2comp_p_96, int ObjectSize_150,
                int SourceSize_151, unsigned int *ele2comp_97,
                unsigned int *eleminwts_98, int ObjectSize_152,
                unsigned int *minwtcomponent_99, unsigned int *partners_100,
                int ObjectSize_153, unsigned int *srcsrc_101,
                int ObjectSize_154, unsigned int *psrc_102,
                unsigned int *edgessrcdst_103, unsigned int *edgessrcwt_104,
                unsigned int *goaheadnodeofcomponent_105, unsigned int
       numNodes_106,
                unsigned int numEdges_107) {
    */

    MST_94 << <gridDim.x, blockDim.x>>>
        (part_outgoing_66 + (0 + (threadIdx.x * tile_64)),
         tile_dyn /*x_size_63*/, tile_dyn, threadOffset,
         OffsetEnd /*SourceSize_165, (p_62 * tile_64)*/, 
         part_ele2comp_67 + (0 + (threadIdx.x * tile_64)), tile_dyn, OffsetEnd,
         ele2comp_51, part_eleminwts_68 + (0 + (threadIdx.x * tile_64)),
         tile_dyn, minwtcomponent_53,
         part_partners_69 + (0 + (threadIdx.x * tile_64)), tile_dyn,
         part_srcsrc_70 + (0 + (threadIdx.x * tile_64)), tile_dyn, psrc_56,
         edgessrcdst_57, edgessrcwt_58, goaheadnodeofcomponent_59, numNodes_60,
         numEdges_61);

    hipDeviceSynchronize();

  } else {

    MST_71(part_outgoing_66 + (0 + (threadIdx.x * 1)), x_size_63,
           SourceSize_165, (p_62 * tile_64), p_62,
           part_ele2comp_67 + (0 + (threadIdx.x * 1)), x_size_63,
           SourceSize_167, ele2comp_51,
           part_eleminwts_68 + (0 + (threadIdx.x * 1)), minwtcomponent_53,
           part_partners_69 + (0 + (threadIdx.x * 1)),
           part_srcsrc_70 + (0 + (threadIdx.x * 1)), psrc_56, edgessrcdst_57,
           edgessrcwt_58, goaheadnodeofcomponent_59, numNodes_60, numEdges_61);

    __syncthreads();
  }
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void MST_25(unsigned int *outgoing_26, int ObjectSize_172,
            unsigned int *ele2comp_p_27, unsigned int *ele2comp_28,
            unsigned int *eleminwts_29, unsigned int *minwtcomponent_30,
            unsigned int *partners_31, unsigned int *srcsrc_32,
            unsigned int *psrc_33, unsigned int *edgessrcdst_34,
            unsigned int *edgessrcwt_35,
            unsigned int *goaheadnodeofcomponent_36, unsigned int numNodes_37,
            unsigned int numEdges_38) {

  int p_39 = TGM_TEMPLATE_0;
  int x_size_40 = ObjectSize_172;
  int tile_41 = ((((x_size_40 + p_39) - 1)) / p_39);

  unsigned int *part_outgoing_43 = outgoing_26;
  unsigned int *part_ele2comp_44 = ele2comp_p_27;
  unsigned int *part_eleminwts_45 = eleminwts_29;
  unsigned int *part_partners_46 = partners_31;
  unsigned int *part_srcsrc_47 = srcsrc_32;

  dim3 dimBlock(TGM_TEMPLATE_1);
  dim3 dimGrid(p_39);
  MST_48 << <dimGrid, dimBlock>>>
      (part_outgoing_43, tile_41, tile_41, x_size_40, part_ele2comp_44, tile_41,
       x_size_40, ele2comp_28, part_eleminwts_45, tile_41, minwtcomponent_30,
       part_partners_46, tile_41, part_srcsrc_47, tile_41, psrc_33,
       edgessrcdst_34, edgessrcwt_35, goaheadnodeofcomponent_36, numNodes_37,
       numEdges_38);
}

template <unsigned int TGM_TEMPLATE_0, unsigned int TGM_TEMPLATE_1>
void MST_tangram(unsigned int *ele2comp_2, unsigned int *eleminwts_3,
                 unsigned int *minwtcomponen_4, unsigned int *partners_5,
                 unsigned int *outgoing_6, unsigned int *srcsrc_7,
                 unsigned int *psrc_8, unsigned int *edgessrcdst_9,
                 unsigned int *edgessrcwt_10,
                 unsigned int *goaheadnodeofcomponent_11,
                 unsigned int numNodes_12, unsigned int numEdges_13) {

  unsigned int *T_ele2comp_14 = ele2comp_2;
  unsigned int *T_ele2comp_p_15 = ele2comp_2;
  unsigned int *T_eleminwts_16 = eleminwts_3;
  unsigned int *T_minwtcomponen_17 = minwtcomponen_4;
  unsigned int *T_partners_18 = partners_5;
  unsigned int *T_outgoing_19 = outgoing_6;
  unsigned int *T_srcsrc_20 = srcsrc_7;
  unsigned int *T_psrc_21 = psrc_8;
  unsigned int *T_edgessrcdst_22 = edgessrcdst_9;
  unsigned int *T_edgessrcwt_23 = edgessrcwt_10;
  unsigned int *T_goaheadnodeofcomponent_24 = goaheadnodeofcomponent_11;

  MST_25<TGM_TEMPLATE_0, TGM_TEMPLATE_1>(
      T_outgoing_19, numNodes_12, T_ele2comp_p_15, T_ele2comp_14,
      T_eleminwts_16, T_minwtcomponen_17, T_partners_18, T_srcsrc_20, T_psrc_21,
      T_edgessrcdst_22, T_edgessrcwt_23, T_goaheadnodeofcomponent_24,
      numNodes_12, numEdges_13);
}

void launch_find_kernel(unsigned int nb, unsigned int nt, unsigned *mstwt,
                        Graph graph, ComponentSpace cs, foru *eleminwts,
                        foru *minwtcomponent, unsigned *partners,
                        unsigned *phore, bool *processinnextiteration,
                        unsigned *goaheadnodeofcomponent, unsigned inpid) {

  MST_tangram<4, 512>(cs.getEle2comp(), (unsigned int *)eleminwts,
                      (unsigned int *)minwtcomponent, partners,
                      graph.getNoutGoing(), graph.getSrcsrc(), graph.getPsrc(),
                      graph.getEdgessrcdst(), graph.getEdgessrcwt(),
                      goaheadnodeofcomponent, graph.getNnodes(),
                      graph.getNedges());
}

/////////////////////////////////////////////////////////////////////////////////////
__global__ void verify_min_elem_child_cdp(
    Graph graph, ComponentSpace cs, unsigned minwt_node, foru minwt,
    /*foru *eleminwts,*/ unsigned *partners, bool *processinnextiteration,
    /*unsigned *goaheadnodeofcomponent, unsigned src,*/ unsigned id,
    /*unsigned srcboss,*/ unsigned degree) {
  // bool minwt_found = false;
  unsigned ii = blockIdx.x * blockDim.x + threadIdx.x;
  if (ii < degree) {
    foru wt = graph.getWeight(minwt_node, ii);
    // printf("%d: looking at %d edge %d wt %d (%d)\n", id, minwt_node, ii, wt,
    // minwt);

    if (wt == minwt) {
      // minwt_found = true;
      unsigned dst = graph.getDestination(minwt_node, ii);
      unsigned tempdstboss = cs.find(dst);
      if (tempdstboss == partners[minwt_node] && tempdstboss != id) {
        processinnextiteration[minwt_node] = true;
        // printf("%d okay!\n", id);
        return;
      }
    }
  } else
    return;
  // printf("component %d is wrong %d - %d - %d, %d\n", id, minwt_found, minwt,
  // ii, degree); // Thread that would printf "okay" should set a verify[id]
  // element. After child kernel finish, if verify[id]!=1, then printf this
  // wrong message
}

__global__ void
verify_min_elem_cdp(unsigned *mstwt, Graph graph, ComponentSpace cs,
                    foru *eleminwts, foru *minwtcomponent, unsigned *partners,
                    unsigned *phore, bool *processinnextiteration,
                    unsigned *goaheadnodeofcomponent, unsigned inpid) {
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
  if (inpid < graph.nnodes)
    id = inpid;

  if (id < graph.nnodes) {

    if (cs.isBoss(id)) {

      if (goaheadnodeofcomponent[id] != graph.nnodes) {
        unsigned minwt_node = goaheadnodeofcomponent[id];
        unsigned degree = graph.getOutDegree(minwt_node);
        foru minwt = minwtcomponent[id];

        if (minwt != MYINFINITY) {
          verify_min_elem_child_cdp
                  << <(int)ceil((float)degree / BLOCK_DIM), BLOCK_DIM>>>
              (graph, cs, minwt_node, minwt, partners, processinnextiteration,
               /*goaheadnodeofcomponent, src,*/ id, /*srcboss,*/ degree);
        }
      }
    }
  }
}

void launch_verify_kernel(unsigned int nb, unsigned int nt, unsigned *mstwt,
                          Graph graph, ComponentSpace cs, foru *eleminwts,
                          foru *minwtcomponent, unsigned *partners,
                          unsigned *phore, bool *processinnextiteration,
                          unsigned *goaheadnodeofcomponent, unsigned inpid) {
  verify_min_elem_cdp << <nb, nt>>>
      (mstwt, graph, cs, eleminwts, minwtcomponent, partners, phore,
       processinnextiteration, goaheadnodeofcomponent, inpid);
}
